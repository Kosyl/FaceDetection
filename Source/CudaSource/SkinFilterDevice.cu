#include "hip/hip_runtime.h"
#include "SkinFilterDevice.h"
#include <cmath>

#define L(x) 105 * log( x + 1 )

__global__ void RGB2IRBKernel( unsigned char *imgIn, float *imgOut, int sizeX, int sizeY )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if( index < sizeX * sizeY )
	{
		const int stride = sizeX * sizeY;
	
		float R = float( imgIn[3 * index    ] );
		float G = float( imgIn[3 * index + 1] );
		float B = float( imgIn[3 * index + 2] );

		imgOut[              index ] = ( L(R) + L(B) + L(G) ) / 3;
		imgOut[ stride     + index ] = L(R) - L(G);
		imgOut[ 2 * stride + index ] = L(B) - ( L(G) + L(R) ) / 2;
	}
}

__global__ void MedianFilterKernel( float *imgIn, float *imgOut, int scale, int *ngb, int sizeX, int sizeY )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	const int regSize = 8;
	
	const int stride = scale * scale;
	float maxVal, minVal;

	if( index < sizeX * sizeY )
	{
		float window[regSize];
		
		const int xPos = index % sizeX;
		const int yPos = (int) index / sizeX;
		
		if( xPos < scale || yPos < scale || xPos > sizeX - scale || yPos > sizeY - scale )
		{
			imgOut[index] = imgIn[index];
			imgOut[index] = 255;
		}
		else
		{
			int ngbIdx = 0;
			float maxVal, minVal;
			int maxIdx, minIdx;
			
			for(int i = 0; i < regSize; i++, ngbIdx += 2) {
				window[i] = imgIn[ xPos + ngb[ngbIdx] + (sizeX * (yPos + ngb[ngbIdx+1])) ];
			}

			while(ngbIdx < 2 * scale * scale) 
			{
				maxVal = -1;
				minVal = 0x7ff0000000000000;
				minIdx = maxIdx = 0;

				for(int i = 0; i < regSize; i++)
				{
					if(window[i] <= minVal)
					{
						minVal = window[i];
						minIdx = i;
					}
					
					if(window[i] >= maxVal)
					{
						maxVal = window[i];
						maxIdx = i;
					}
				}

				window[minIdx] = imgIn[ xPos + ngb[ngbIdx] + (sizeX * (yPos + ngb[ngbIdx+1]))];
				ngbIdx += 2;
				if(ngbIdx >= 2 * scale * scale )
					break;
				window[maxIdx] = imgIn[ xPos + ngb[ngbIdx] + (sizeX * (yPos + ngb[ngbIdx+1]))];
				ngbIdx += 2;
				if(ngbIdx >= 2 * scale * scale )
					break;
			}

			for( int x=0; x < regSize; x++ )
			{
				for( int y = 0; y < regSize - 1; y++ )
				{
					if(window[y] > window[y+1])
					{
						float temp = window[y+1];
						window[y+1] = window[y];
						window[y] = temp;
					}
				}
			}

			imgOut[ index ] = ( window[4] ) ;
		}
	}
}

/*__global__ void MedianFilterKernel( float *imgIn, float *imgOut, int scale, int *ngb, int sizeX, int sizeY )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	scale = 7;

	if( index < sizeX * sizeY )
	{
		float window[1000];
		
		const int xPos = index % sizeX;
		const int yPos = (int) index / sizeX;
		
		if( xPos < scale || yPos < scale || xPos > sizeX - scale || yPos > sizeY - scale )
		{
			imgOut[index] = imgIn[index];
			imgOut[index] = 255;
		}
		else
		{
			int windCnt = 0;
			for( int i = -scale; i <= scale; i++ )
			{
				for( int j = -scale; j <= scale; j++ )
				{
					const float val = imgIn[ xPos + i + (sizeX * (yPos + j)) ];
					window[windCnt] = val;
					windCnt++;
				
				}
			}
			
			// int windCnt = 0;
			// for( int k = 0; k < 2 * scale * scale; k += 2 )
			// {
			// 	const float val = imgIn[ xPos + ngb[k] + (sizeX * (yPos + ngb[k+1])) ];
			//	window[windCnt] = val;
			//	windCnt++;
			// }
			
			float median = GetMedianDevice( window, scale );
			imgOut[index] = median;
			// imgOut[index] = imgIn[ xPos + (sizeX * (yPos)) ];
		}
	}
}*/

__device__ float GetMedianDevice( float *img, int scale )
{
	for( int x=0; x < scale * scale; x++ )
	{
		for( int y = 0; y < (scale * scale) - 1; y++ )
		{
			if(img[y] > img[y+1])
			{
				float temp = img[y+1];
				img[y+1] = img[y];
				img[y] = temp;
			}
		}
	}

	return img[ (((scale * scale) - 1 ) / 2 )+ 1 ];
}

__global__ void HueSaturationKernel( float *hue, float *saturation, float *By, float *Rb, int sizeX, int sizeY )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if( index < sizeX * sizeY )
	{
		const float  Rg_ = By[index];
		const float  By_ = Rb[index];

		hue[index] = (atan2f(Rg_, By_) / 3.14) * 180;
		saturation[index] = hypotf(Rg_, By_);
	}
}

void TextureKernel( float *texture, float *imgIn, float *imgFiltered, int sizeX, int sizeY )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if( index < sizeX * sizeY )
	{
		texture[index] = abs( imgIn[index] - imgFiltered[index] );
	}
}

void GenerateMapKernel( float *medianImg, float *hue, float *saturation, unsigned char *map, int sizeX, int sizeY )
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if( index < sizeX * sizeY )
	{
	
		bool m = medianImg[index] < 4.5;
		bool h = hue[index] > 120 && hue[index] < 160;
		bool s = saturation[index] > 10 && saturation[index] < 60;

		if( m && h && s )
		{
			map[index] = 255;
			return;
		}

		m = medianImg[index] < 4.5;
		h = hue[index] > 150 && hue[index] < 180;
		s = saturation[index] > 20 && saturation[index] < 80;

		if( m && h && s )
		{
			map[index] = 255;
			return;
		}

		map[index] = 0;
	}
}

